#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <iostream>

__global__ void vectorAdd(int A[BLOCK_SIZE][BLOCK_SIZE], int B[BLOCK_SIZE][BLOCK_SIZE], int C[BLOCK_SIZE][BLOCK_SIZE]){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row >= h || col >= w){
        return;
    }
    C[row][col] = A[row][col] + B[row][col];
}

int main(){
    int d_A[BLOCK_SIZE][BLOCK_SIZE];
    int d_B[BLOCK_SIZE][BLOCK_SIZE];
    int d_C[BLOCK_SIZE][BLOCK_SIZE];

    int C[BLOCK_SIZE][BLOCK_SIZE];

    for(int i = 0; i < BLOCK_SIZE; i++){
        for(int j = 0; j < BLOCK_SIZE; j++){
            d_A[i][j] = i + j;
            d_B[i][j] = i + j;
        }
    }

    const dim3 BLOCK_SIZE = dim3(3, 1, 1);
    const dim3 GRID_SIZE = dim3(2, 2, 1);

    vectorAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, BLOCK_SIZE * BLOCK_SIZE, hipMemcpyDeviceToHost);

    for(int i = 0; i < BLOCK_SIZE; i++){
        for(int j = 0; j < BLOCK_SIZE; j++){
            printf("%d\n", C[i][j]);
        }
    }
}