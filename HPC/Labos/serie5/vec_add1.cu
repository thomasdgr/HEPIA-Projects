
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <iostream>

// COURS

// CUDA kernel pour l'addition de vecteurs (__global__ est utilisé pour dire que c'est une fonction kernel)
// -> on aura un thread pour chaque élément dans "c"
__global__ void vectorAdd(const int* a, const int* b, int* c, int N) {
  // Thread ID globale
  int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
  // blockIdx.x -> contient l'id du bloc actuel qui execute le code (.x car c'est une grille 1D)
  // Certains ne doivent rien faire !
  if (tid < N) c[tid] = a[tid] + b[tid];
}

int main() {
  // la taille est une puissance de deux, cela est simplement pour assurer
  // les divisions entières par 1024 threads par block (c'est juste du confort)
  const int N = 1 << 28;
  const size_t n_bytes = sizeof(int)*N;

  //std::cout << "Number of integers: " << N << std::endl;
  //std::cout << "Vector size: " << (double) n_bytes/1000000000.0 << " [GB]" << std::endl;

  // les trois vecteurs: C = A + B, ce sont les vecteurs du host (CPU)
  int* h_a = (int*) malloc(n_bytes);
  int* h_b = (int*) malloc(n_bytes);
  int* h_c = (int*) malloc(n_bytes);

  // Du random: 0 à 99 sur les vecteurs du host qui seront copiés sur le device
  // on copie A et B sur le device qui va nous retourner C tel que C = A copié + B copié
  for (int i = 0; i < N; i++) {
    h_a[i] = rand() % 100;
    h_b[i] = rand() % 100;
  }

  // allocation sur le device
  int* d_a; 
  int* d_b;
  int* d_c;
  // allocation de la mémoire sur le gpu -> device
  hipMalloc(&d_a, n_bytes); // stock les entrées
  hipMalloc(&d_b, n_bytes); // stock les entrées
  hipMalloc(&d_c, n_bytes); // stock le résultat

  // Host -> Device
  // on copie h_a dans d_a et h_b dans d_b
  hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice);

  // Threads par block (max 1024),
  // C'est la taille du block
  int NUM_THREADS = 1024;

  // Block par Grid: il faut au moins autant de threads que d'éléments.
  // Mais on a que 1024 threads par block (max), donc on n'ajoute un block
  // en plus si nécessaire (i.e. padding).
  // C'est la taille de la grille
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  // On lance le Kernel (il faut utiliser les chevrons): on note que c'est asynchrone
  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

  // Device -> Host: on rammène C et on note que cudaMemcpy est synchrone
  hipMemcpy(h_c, d_c, n_bytes, hipMemcpyDeviceToHost);

  // Tout est ok ?
  for (int i = 0; i < N; i++) {
    assert(h_c[i] == h_a[i] + h_b[i]);
  }

  // On libère la mémoire
  free(h_a);
  free(h_b);
  free(h_c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  //std::cout << "Addition de vecteurs: OK" << std::endl;

  return 0;
}