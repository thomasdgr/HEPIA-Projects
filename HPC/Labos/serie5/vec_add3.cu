

#include "stdio.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// DOC BAOBAB
 
// Handle CUDA error messages
inline void CUDA_HandleError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess)
  {
    std::cout << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
    exit(EXIT_FAILURE);
  }
}
#define CUDA_HANDLE_ERROR( err ) (CUDA_HandleError( err, __FILE__, __LINE__ ))
 
// fonction kernel s'executant sur le device
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}
 
// affichage d'un vecteur
void printVec(std::vector<float> &vec){
    for(auto e : vec) std::cout << e << ", ";
    std::cout << std::endl;
}
 
// code host
int main()
{
  int N = 10;
  size_t size = N * sizeof(float);
 
  // allocation de la memoire sur le host
  std::vector<float> h_A(N, 1.0);
  std::vector<float> h_B(N, 2.0);
  std::vector<float> h_C(N, -1.0);
 
  std::cout << "h_C avant execution kernel : " << std::endl;
  printVec(h_C);
 
  // allocation de la memoire sur le device
  float* d_A;
  CUDA_HANDLE_ERROR( hipMalloc(&d_A, size) );
  float* d_B;
  hipMalloc(&d_B, size);
  float* d_C;
  hipMalloc(&d_C, size);
 
  // poie des vecteur h_A et h_B sur les espaces memoire du device d_A et d_B
  hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);
 
  // preparation de la configuration d'execution
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  // execution du kernel
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  // attendre que le kernel se termine et recuperer la derniere erreur
  CUDA_HANDLE_ERROR( hipPeekAtLastError() );
  CUDA_HANDLE_ERROR( hipDeviceSynchronize() );
 
  // copie des donnees du device vers le host (d_C vers h_C)
  hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);
 
  std::cout << "h_C apres execution kernel : " << std::endl;
  printVec(h_C);
 
  // liberation de la memoire sur le device
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}