
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <iostream>

__global__ void laplace(float* d_in, int w, int h, float* d_out){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if(row < h - 1 && row > 0 && col < w - 1 && col > 0){
    d_out[row * w + col] = 0.25 * (
      d_in[(row - 1) * w + col] + d_in[(row + 1) * w + col] +
      d_in[row * w + (col - 1)] + d_in[row * w + (col + 1)] );
  }
}

float* heat_solver(int N, int M, int T, float* north, float* south, float* east, float* west, dim3 grid_dim, dim3 block_dim){
  const size_t n_bytes = sizeof(float) * (N * M);

  float *d = (float*) malloc(n_bytes);
  
  for(int k = 0; k < M; k++){
    d[k] = north[k];
    d[(N-1)*M + k] = south[k];
  }

  for(int k = 0; k < N-2; k++){
    d[(k+1)*M] = west[k];
    d[(k+2)*M - 1] = east[k];
  }

  float* d_in;
  hipMalloc(&d_in, n_bytes);
  hipMemcpy(d_in, d, n_bytes, hipMemcpyHostToDevice);

  float* d_out;
  hipMalloc(&d_out, n_bytes);
  hipMemcpy(d_out, d, n_bytes, hipMemcpyHostToDevice);

  for(int i = 0; i < T; i++){
    if(i % 2 == 0){
      laplace<<<grid_dim, block_dim>>>(d_in, N, M, d_out);
    } else {
      laplace<<<grid_dim, block_dim>>>(d_out, N, M, d_in);
    }
    hipDeviceSynchronize();
  }  
  
  // faut verifier que je fais pas de la merde ici
  if(T % 2 == 0){
    hipMemcpy(d, d_out, n_bytes, hipMemcpyDeviceToHost);
  } else {
    hipMemcpy(d, d_in, n_bytes, hipMemcpyDeviceToHost);
  }
   
  return d;
}