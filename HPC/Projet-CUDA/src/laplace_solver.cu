
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
   }
}

__global__ void laplace(float* d_in, int w, int h, float* d_out){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if( (row * w + col) < w * h){
    if((row < h - 1) && (row > 0) && (col < w - 1) && (col > 0)){
      d_out[row * w + col] = 0.25 * (
        d_in[(row - 1) * w + col] + d_in[(row + 1) * w + col] +
        d_in[row * w + (col - 1)] + d_in[row * w + (col + 1)] );
    }
  }
}

float* heat_solver(int N, int M, int T, float* north, float* south, float* east, float* west, dim3 grid_dim, dim3 block_dim){
  const size_t n_bytes = sizeof(float) * (N * M);

  float *d = (float*) calloc(n_bytes, sizeof(float));
  for(int k = 0; k < M; k++){
    d[k] = north[k];
    d[(N-1)*M + k] = south[k];
  }

  for(int k = 0; k < N-2; k++){
    d[(k+1)*M] = west[k];
    d[(k+2)*M - 1] = east[k];
  }

  float* d_in;
  gpuErrchk(hipMalloc(&d_in, n_bytes));
  gpuErrchk(hipMemcpy(d_in, d, n_bytes, hipMemcpyHostToDevice));

  float* d_out;
  gpuErrchk(hipMalloc(&d_out, n_bytes));
  gpuErrchk(hipMemcpy(d_out, d, n_bytes, hipMemcpyHostToDevice));

  for(int i = 0; i < T; i++){
    if(i % 2 == 0){
      laplace<<<grid_dim, block_dim>>>(d_in, M, N, d_out);
    } else {
      laplace<<<grid_dim, block_dim>>>(d_out, M, N, d_in);
    }
    gpuErrchk(hipDeviceSynchronize());
  }  
  
  if(T % 2 == 0){
    gpuErrchk(hipMemcpy(d, d_out, n_bytes, hipMemcpyDeviceToHost));
  } else {
    gpuErrchk(hipMemcpy(d, d_in, n_bytes, hipMemcpyDeviceToHost));
  }

  gpuErrchk(hipFree(d_in));
  gpuErrchk(hipFree(d_out));
   
  return d;
}