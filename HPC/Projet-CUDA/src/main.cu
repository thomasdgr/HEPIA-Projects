#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <time.h>
#include "laplace_solver.cuh"

int main(int argc, const char * argv[]){
  const size_t N = atoi(argv[1]), M = atoi(argv[2]);
  float *north = (float *)malloc(sizeof(float) * M);
  float *south = (float *)malloc(sizeof(float) * M);
  float *east = (float *)malloc(sizeof(float) * (N - 2));
  float *west = (float *)malloc(sizeof(float) * (N - 2));
  
  for (int k = 0; k < M; k++) {
    north[k] = 1.0;
    south[k] = 2.0;
  }

  for (int k = 0; k < N-2; k++) {    
    west[k] = 3.0;
    east[k] = 4.0;
  }

  clock_t c = clock();
  float *heated = heat_solver(N, M, atoi(argv[3]), north, south, east, west, dim3(atoi(argv[4]), atoi(argv[5]), 1), dim3(atoi(argv[6]), atoi(argv[7]), 1));
  printf("N=%d - M=%d\nT=%d\nTime=%f\n", N,M,atoi(argv[3]), (double)(clock() - c) / CLOCKS_PER_SEC);

  /*for(int i = 0; i < N; i++){
    for(int j = 0; j < M; j++){
      printf("%f\t", heated[i * M + j]);
    }
    printf("\n");
  }*/

  // Test: N & S
  for (int k = 0; k < M; k++) {
    assert(heated[k] == north[k]);
    assert(heated[(N-1)*M + k] == south[k]);
  }

  // Test: W & E
  for (int k = 0; k < N-2; k++) {
    assert(heated[(k+1)*M] == west[k]);
    assert(heated[(k+2)*M - 1] == east[k]);
  }

  return 0;
}
